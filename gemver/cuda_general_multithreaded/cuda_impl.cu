#include "hip/hip_runtime.h"
#include <gemver.h>
#include <polybench.h>
#include <stdio.h>

#include <cuda_support.cuh>

void initialise_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                          DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                          DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                          DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)n;
    (void)alpha;
    (void)beta;
    (void)A;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)w;
    (void)x;
    (void)y;
    (void)z;
}

void finish_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                      DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                      DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                      DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)n;
    (void)alpha;
    (void)beta;
    (void)A;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)w;
    (void)x;
    (void)y;
    (void)z;
}

__global__ void kernel_1(int n, DATA_TYPE *A, DATA_TYPE *u1, DATA_TYPE *v1, DATA_TYPE *u2, DATA_TYPE *v2) {
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < n; i += blockDim.y * gridDim.y) {
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x) {
            A[i * n + j] = A[i * n + j] + u1[i] * v1[j] + u2[i] * v2[j];
        }
    }
}

__global__ void kernel_2(int n, const DATA_TYPE beta, const DATA_TYPE *A, DATA_TYPE *x, const DATA_TYPE *y,
                         const DATA_TYPE *z) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < _PB_N; i += gridDim.x * blockDim.x) {
        for (int j = 0; j < _PB_N; j++) {
            // could tile accesses to A so they're not done as strided.
            x[i] = x[i] + beta * A[j * n + i] * y[j];
        }
        x[i] = x[i] + z[i];
    }
}

__global__ void kernel_3(const int n, const DATA_TYPE alpha, const DATA_TYPE *A, DATA_TYPE *w, const DATA_TYPE *x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < _PB_N; i += gridDim.x * blockDim.x) {
        for (int j = 0; j < _PB_N; j++) {
            w[i] = w[i] + alpha * A[i * n + j] * x[j];
        }
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void kernel_gemver(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                   DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                   DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                   DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                   DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    DATA_TYPE *A_dev = NULL;
    DATA_TYPE *u1_dev = NULL;
    DATA_TYPE *v1_dev = NULL;
    DATA_TYPE *u2_dev = NULL;
    DATA_TYPE *v2_dev = NULL;
    DATA_TYPE *w_dev = NULL;
    DATA_TYPE *x_dev = NULL;
    DATA_TYPE *y_dev = NULL;
    DATA_TYPE *z_dev = NULL;

    gpuErrchk(hipMalloc(&A_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&u1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&v1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&u2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&v2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&w_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&x_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&y_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&z_dev, sizeof(DATA_TYPE) * n));

    gpuErrchk(hipMemcpy(A_dev, A, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(u1_dev, u1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v1_dev, v1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(u2_dev, u2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(v2_dev, v2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(w_dev, w, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(x_dev, x, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(y_dev, y, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(z_dev, z, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));

    dim3 tpb1(8, 8);
    dim3 bpg1(1, 2);
    dim3 tpb2(64);
    dim3 bpg2(4);
    dim3 tpb3(64);
    dim3 bpg3(4);
    kernel_1<<<bpg1, tpb1>>>(n, A_dev, u1_dev, v1_dev, u2_dev, v2_dev);
    kernel_2<<<bpg2, tpb2>>>(n, beta, A_dev, x_dev, y_dev, z_dev);
    kernel_3<<<bpg3, tpb3>>>(n, alpha, A_dev, w_dev, x_dev);

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(w, w_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(A, A_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(x, x_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(A_dev));
    gpuErrchk(hipFree(u1_dev));
    gpuErrchk(hipFree(v1_dev));
    gpuErrchk(hipFree(u2_dev));
    gpuErrchk(hipFree(v2_dev));
    gpuErrchk(hipFree(w_dev));
    gpuErrchk(hipFree(x_dev));
    gpuErrchk(hipFree(y_dev));
    gpuErrchk(hipFree(z_dev));
}