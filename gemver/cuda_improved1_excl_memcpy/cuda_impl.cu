#include "hip/hip_runtime.h"
#include <assert.h>
#include <gemver.h>
#include <polybench.h>
#include <stdio.h>

#include <cuda_support.cuh>

typedef struct {
    DATA_TYPE *A_dev;
    DATA_TYPE *u1_dev;
    DATA_TYPE *v1_dev;
    DATA_TYPE *u2_dev;
    DATA_TYPE *v2_dev;
    DATA_TYPE *w_dev;
    DATA_TYPE *x_dev;
    DATA_TYPE *y_dev;
    DATA_TYPE *z_dev;
    dim3 tpb_Ax;
    dim3 bpg_Ax;
    dim3 tpb_3;
    dim3 bpg_3;
} kernel_init_t;

kernel_init_t device_addrs;

void initialise_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                          DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                          DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                          DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)alpha;
    (void)beta;
    gpuErrchk(hipMalloc(&device_addrs.A_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.u1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.v1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.u2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.v2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.w_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.x_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.y_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.z_dev, sizeof(DATA_TYPE) * n));

    hipMemcpy(device_addrs.A_dev, A, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.u1_dev, u1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.v1_dev, v1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.u2_dev, u2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.v2_dev, v2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.w_dev, w, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.x_dev, x, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.y_dev, y, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.z_dev, z, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int sm_count = deviceProp.multiProcessorCount;
    int max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;
    int max_threads_for_gpu = sm_count * max_threads_per_sm;

#ifndef TPB
    unsigned int tpb = 64;
#else
    unsigned int tpb = TPB;
#endif

    device_addrs.tpb_Ax.x = tpb;
    device_addrs.bpg_Ax.x = 64 * max_threads_for_gpu / device_addrs.tpb_Ax.x;

    device_addrs.tpb_3.x = tpb;
    device_addrs.bpg_3.x = max_threads_for_gpu / device_addrs.tpb_3.x;
}

void finish_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                      DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                      DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                      DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)n;
    (void)alpha;
    (void)beta;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)y;
    (void)z;

    hipMemcpy(w, device_addrs.w_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);
    hipMemcpy(A, device_addrs.A_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(x, device_addrs.x_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);

    hipFree(device_addrs.A_dev);
    hipFree(device_addrs.u1_dev);
    hipFree(device_addrs.v1_dev);
    hipFree(device_addrs.u2_dev);
    hipFree(device_addrs.v2_dev);
    hipFree(device_addrs.w_dev);
    hipFree(device_addrs.x_dev);
    hipFree(device_addrs.y_dev);
    hipFree(device_addrs.z_dev);
}

__global__ void kernel_Ax_combi(int n, const DATA_TYPE beta, DATA_TYPE *A, DATA_TYPE *u1, DATA_TYPE *v1, DATA_TYPE *u2,
                                DATA_TYPE *v2, DATA_TYPE *x, const DATA_TYPE *y, const DATA_TYPE *z

) {
    extern __shared__ DATA_TYPE all_shared[];
    DATA_TYPE *u1_s = all_shared;
    DATA_TYPE *u2_s = &all_shared[n];
    DATA_TYPE *y_s = &all_shared[2 * n];
    // note: shared memory is per-block, NOT grid-wide
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        u1_s[i] = u1[i];
        u2_s[i] = u2[i];
        y_s[i] = y[i];
    }
    __syncthreads();
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        DATA_TYPE x_loc = x[i];
        DATA_TYPE v1_loc = v1[i];
        DATA_TYPE v2_loc = v2[i];
        for (int j = 0; j < n; j++) {
            double A_loc = A[j * n + i];
            A_loc += u1_s[j] * v1_loc + u2_s[j] * v2_loc;
            x_loc = x_loc + beta * A_loc * y_s[j];
            A[j * n + i] = A_loc;
        }
        x[i] = x_loc + z[i];
    }
}

__global__ void kernel_3(const int n, const DATA_TYPE alpha, const DATA_TYPE *A, DATA_TYPE *w, const DATA_TYPE *x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < _PB_N; i += gridDim.x * blockDim.x) {
        DATA_TYPE loc = w[i];
        for (int j = 0; j < _PB_N; j++) {
            int jmod = (j + threadIdx.x % 32) % _PB_N;
            loc += alpha * A[i * n + jmod] * x[jmod];
        }
        w[i] = loc;
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void kernel_gemver(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                   DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                   DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                   DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                   DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    kernel_Ax_combi<<<device_addrs.bpg_Ax, device_addrs.tpb_Ax, 3 * n * sizeof(DATA_TYPE)>>>(
        n, beta, device_addrs.A_dev, device_addrs.u1_dev, device_addrs.v1_dev, device_addrs.u2_dev, device_addrs.v2_dev,
        device_addrs.x_dev, device_addrs.y_dev, device_addrs.z_dev);
    kernel_3<<<device_addrs.bpg_3, device_addrs.tpb_3>>>(n, alpha, device_addrs.A_dev, device_addrs.w_dev,
                                                         device_addrs.x_dev);

    gpuErrchk(hipDeviceSynchronize());
}