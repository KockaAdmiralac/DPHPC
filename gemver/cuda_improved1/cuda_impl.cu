#include "hip/hip_runtime.h"
#include <assert.h>
#include <gemver.h>
#include <polybench.h>
#include <stdio.h>

#include <cuda_support.cuh>

void initialise_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                          DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                          DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                          DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)n;
    (void)alpha;
    (void)beta;
    (void)A;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)w;
    (void)x;
    (void)y;
    (void)z;
}

void finish_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                      DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                      DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                      DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)n;
    (void)alpha;
    (void)beta;
    (void)A;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)w;
    (void)x;
    (void)y;
    (void)z;
}

__global__ void kernel_1(int n, DATA_TYPE *A, DATA_TYPE *u1, DATA_TYPE *v1, DATA_TYPE *u2, DATA_TYPE *v2) {
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x) {
        for (int i = 0; i < n; i++) {
            A[i * n + j] = A[i * n + j] + u1[i] * v1[j] + u2[i] * v2[j];
        }
    }
}

__global__ void kernel_2(int n, const DATA_TYPE beta, const DATA_TYPE *A, DATA_TYPE *x, const DATA_TYPE *y,
                         const DATA_TYPE *z) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < _PB_N; i += gridDim.x * blockDim.x) {
        for (int j = 0; j < _PB_N; j++) {
            // could tile accesses to A so they're not done as strided.
            x[i] = x[i] + beta * A[j * n + i] * y[j];
        }
        x[i] = x[i] + z[i];
    }
}

__global__ void kernel_Ax_combi(int n, const DATA_TYPE beta, DATA_TYPE *A, DATA_TYPE *u1, DATA_TYPE *v1, DATA_TYPE *u2,
                                DATA_TYPE *v2, DATA_TYPE *x, const DATA_TYPE *y, const DATA_TYPE *z

) {
    extern __shared__ DATA_TYPE all_shared[];
    DATA_TYPE *u1_s = all_shared;
    DATA_TYPE *u2_s = &all_shared[n];
    DATA_TYPE *y_s = &all_shared[2 * n];
    // note: shared memory is per-block, NOT grid-wide
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        u1_s[i] = u1[i];
        u2_s[i] = u2[i];
        y_s[i] = y[i];
    }
    __syncthreads();
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        DATA_TYPE x_loc = x[i];
        DATA_TYPE v1_loc = v1[i];
        DATA_TYPE v2_loc = v2[i];
        for (int j = 0; j < n; j++) {
            double A_loc = A[j * n + i];
            A_loc += u1_s[j] * v1_loc + u2_s[j] * v2_loc;
            x_loc = x_loc + beta * A_loc * y_s[j];
            A[j * n + i] = A_loc;
        }
        x[i] = x_loc + z[i];
    }
}

__global__ void kernel_3(const int n, const DATA_TYPE alpha, const DATA_TYPE *A, DATA_TYPE *w, const DATA_TYPE *x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < _PB_N; i += gridDim.x * blockDim.x) {
        DATA_TYPE loc = w[i];
        for (int j = 0; j < _PB_N; j++) {
            int jmod = (j + threadIdx.x % 32) % _PB_N;
            loc += alpha * A[i * n + jmod] * x[jmod];
        }
        w[i] = loc;
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void kernel_gemver(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                   DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                   DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                   DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                   DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    DATA_TYPE *A_dev = NULL;
    DATA_TYPE *u1_dev = NULL;
    DATA_TYPE *v1_dev = NULL;
    DATA_TYPE *u2_dev = NULL;
    DATA_TYPE *v2_dev = NULL;
    DATA_TYPE *w_dev = NULL;
    DATA_TYPE *x_dev = NULL;
    DATA_TYPE *y_dev = NULL;
    DATA_TYPE *z_dev = NULL;

    gpuErrchk(hipMalloc(&A_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&u1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&v1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&u2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&v2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&w_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&x_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&y_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&z_dev, sizeof(DATA_TYPE) * n));

    hipMemcpy(A_dev, A, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(u1_dev, u1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(v1_dev, v1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(u2_dev, u2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(v2_dev, v2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(w_dev, w, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(x_dev, x, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(y_dev, y, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(z_dev, z, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int sm_count = deviceProp.multiProcessorCount;
    int max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;
    int max_threads_for_gpu = sm_count * max_threads_per_sm;

    dim3 tpb1(64);
    int blocks_k1 = max_threads_for_gpu / tpb1.x;
    dim3 bpg1(blocks_k1);
    // printf("Using %d blocks_k1, max threads for GPU is %d, max threads per SM is %d\n", blocks_k1,
    // max_threads_for_gpu, max_threads_per_sm);

    dim3 tpb2(64);
    dim3 bpg2(max_threads_for_gpu / tpb2.x);
    dim3 tpb3(64);
    dim3 bpg3(max_threads_for_gpu / tpb3.x);
    kernel_Ax_combi<<<bpg1, tpb1, 3 * n * sizeof(DATA_TYPE)>>>(n, beta, A_dev, u1_dev, v1_dev, u2_dev, v2_dev, x_dev,
                                                               y_dev, z_dev);
    // kernel_1<<<bpg1, tpb1>>>(n, A_dev, u1_dev, v1_dev, u2_dev, v2_dev);
    // kernel_2<<<bpg2, tpb2>>>(n, beta, A_dev, x_dev, y_dev, z_dev);
    kernel_3<<<bpg3, tpb3>>>(n, alpha, A_dev, w_dev, x_dev);

    hipDeviceSynchronize();

    hipMemcpy(w, w_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);
    hipMemcpy(A, A_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(x, x_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);

    hipFree(A_dev);
    hipFree(u1_dev);
    hipFree(v1_dev);
    hipFree(u2_dev);
    hipFree(v2_dev);
    hipFree(w_dev);
    hipFree(x_dev);
    hipFree(y_dev);
    hipFree(z_dev);
}