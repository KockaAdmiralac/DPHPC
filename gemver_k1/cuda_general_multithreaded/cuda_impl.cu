#include "hip/hip_runtime.h"
#include <gemver.h>
#include <polybench.h>
#include <stdio.h>

#include <cuda_support.cuh>

typedef struct {
    DATA_TYPE *A_dev;
    DATA_TYPE *u1_dev;
    DATA_TYPE *v1_dev;
    DATA_TYPE *u2_dev;
    DATA_TYPE *v2_dev;
    DATA_TYPE *w_dev;
    DATA_TYPE *x_dev;
    DATA_TYPE *y_dev;
    DATA_TYPE *z_dev;
    dim3 tpb_1;
    dim3 bpg_1;
    dim3 tpb_2;
    dim3 bpg_2;
    dim3 tpb_3;
    dim3 bpg_3;
} kernel_init_t;

kernel_init_t device_addrs;
void initialise_benchmark(int argc, char **argv, int n, DATA_TYPE *alpha, DATA_TYPE *beta,
                          DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n), DATA_TYPE POLYBENCH_1D(u1, N2, n),
                          DATA_TYPE POLYBENCH_1D(v1, N2, n), DATA_TYPE POLYBENCH_1D(u2, N2, n),
                          DATA_TYPE POLYBENCH_1D(v2, N2, n), DATA_TYPE POLYBENCH_1D(w, N2, n),
                          DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                          DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)alpha;
    (void)beta;
    (void)argc;
    (void)argv;
    gpuErrchk(hipMalloc(&device_addrs.A_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.u1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.v1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.u2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.v2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.w_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.x_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.y_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.z_dev, sizeof(DATA_TYPE) * n));

    hipMemcpy(device_addrs.A_dev, A, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.u1_dev, u1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.v1_dev, v1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.u2_dev, u2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.v2_dev, v2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.w_dev, w, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.x_dev, x, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.y_dev, y, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(device_addrs.z_dev, z, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int sm_count = deviceProp.multiProcessorCount;
    int max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;
    int max_threads_for_gpu = sm_count * max_threads_per_sm;

#ifndef TPB
    unsigned int tpb = 64;
#else
    unsigned int tpb = TPB;
#endif

    device_addrs.tpb_1.x = tpb;
    device_addrs.bpg_1.x = max_threads_for_gpu / device_addrs.tpb_1.x;

    device_addrs.tpb_2.x = tpb;
    device_addrs.bpg_2.x = max_threads_for_gpu / device_addrs.tpb_2.x;

    device_addrs.tpb_3.x = tpb;
    device_addrs.bpg_3.x = max_threads_for_gpu / device_addrs.tpb_3.x;

    *alpha = 1.5;
    *beta = 1.2;

    DATA_TYPE fn = (DATA_TYPE)n;

    for (int i = 0; i < n; i++) {
        u1[i] = i;
        u2[i] = ((i + 1) / fn) / 2.0;
        v1[i] = ((i + 1) / fn) / 4.0;
        v2[i] = ((i + 1) / fn) / 6.0;
        y[i] = ((i + 1) / fn) / 8.0;
        z[i] = ((i + 1) / fn) / 9.0;
        x[i] = 0.0;
        w[i] = 0.0;
        for (int j = 0; j < n; j++) A[i][j] = (DATA_TYPE)(i * j % n) / n;
    }

    gpuErrchk(hipMalloc(&device_addrs.A_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.u1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.v1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.u2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.v2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.w_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.x_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.y_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&device_addrs.z_dev, sizeof(DATA_TYPE) * n));

    gpuErrchk(hipMemcpy(device_addrs.A_dev, A, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.u1_dev, u1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.v1_dev, v1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.u2_dev, u2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.v2_dev, v2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.w_dev, w, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.x_dev, x, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.y_dev, y, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.z_dev, z, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice));

    gpuErrchk(hipDeviceSynchronize());
}

void finish_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                      DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                      DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                      DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)n;
    (void)alpha;
    (void)beta;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)y;
    (void)z;

    hipMemcpy(w, device_addrs.w_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);
    hipMemcpy(A, device_addrs.A_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(x, device_addrs.x_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);

    hipFree(device_addrs.A_dev);
    hipFree(device_addrs.u1_dev);
    hipFree(device_addrs.v1_dev);
    hipFree(device_addrs.u2_dev);
    hipFree(device_addrs.v2_dev);
    hipFree(device_addrs.w_dev);
    hipFree(device_addrs.x_dev);
    hipFree(device_addrs.y_dev);
    hipFree(device_addrs.z_dev);
}

__global__ void kernel_1(int n, DATA_TYPE *A, DATA_TYPE *u1, DATA_TYPE *v1, DATA_TYPE *u2, DATA_TYPE *v2) {
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < n; i += blockDim.y * gridDim.y) {
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x) {
            A[i * n + j] = A[i * n + j] + u1[i] * v1[j] + u2[i] * v2[j];
        }
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void kernel_gemver(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                   DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                   DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                   DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                   DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    kernel_1<<<device_addrs.bpg_1, device_addrs.tpb_1>>>(n, device_addrs.A_dev, device_addrs.u1_dev,
                                                         device_addrs.v1_dev, device_addrs.u2_dev, device_addrs.v2_dev);

    gpuErrchk(hipDeviceSynchronize());
}
