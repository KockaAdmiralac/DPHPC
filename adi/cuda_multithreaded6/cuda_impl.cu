#include "hip/hip_runtime.h"
/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include <cuda_support.cuh>

#include "adi.h"

typedef struct {
    DATA_TYPE *u_dev;
    DATA_TYPE *v_dev;
    DATA_TYPE *p_dev;
    DATA_TYPE *q_dev;
} kernel_init_t;

kernel_init_t device_addrs;

void transpose_in_place(DATA_TYPE POLYBENCH_2D(arr, N2, N2, n, n)) {
    for (int i = 0; i < N2; i++) {
        for (int j = i + 1; j < N2; j++) {
            DATA_TYPE upper_right = arr[i][j];
            DATA_TYPE lower_left = arr[j][i];
            arr[j][i] = upper_right;
            arr[i][j] = lower_left;
        }
    }
}

void copy_array_transposed(DATA_TYPE POLYBENCH_2D(arr, N2, N2, n, n), void *device_addr, hipMemcpyKind dir) {
    if (dir == hipMemcpyHostToDevice) {
        transpose_in_place(arr);
        gpuErrchk(hipMemcpy(device_addr, arr, sizeof(DATA_TYPE) * N2 * N2, hipMemcpyHostToDevice));
    } else {
        gpuErrchk(hipMemcpy(arr, device_addr, sizeof(DATA_TYPE) * N2 * N2, hipMemcpyDeviceToHost));
        gpuErrchk(hipDeviceSynchronize());
        transpose_in_place(arr);
    }
}

void initialise_benchmark(int argc, char **argv, int tsteps, int n, DATA_TYPE POLYBENCH_2D(u, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_2D(v, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(p, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_2D(q, N2, N2, n, n)) {
    (void)tsteps;
    (void)n;

    int i, j;

    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            u[i][j] = (DATA_TYPE)(i + n - j) / n;
        }
    gpuErrchk(hipMalloc(&device_addrs.u_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.v_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.p_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.q_dev, sizeof(DATA_TYPE) * n * n));

    gpuErrchk(hipMemcpy(device_addrs.u_dev, u, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    copy_array_transposed(v, device_addrs.v_dev, hipMemcpyHostToDevice);
    copy_array_transposed(p, device_addrs.p_dev, hipMemcpyHostToDevice);
    copy_array_transposed(q, device_addrs.q_dev, hipMemcpyHostToDevice);
}

void finish_benchmark(int tsteps, int n, DATA_TYPE POLYBENCH_2D(u, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_2D(v, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(p, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_2D(q, N2, N2, n, n)) {
    (void)tsteps;
    (void)n;

    gpuErrchk(hipMemcpy(u, device_addrs.u_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));
    copy_array_transposed(v, device_addrs.v_dev, hipMemcpyDeviceToHost);
    copy_array_transposed(p, device_addrs.p_dev, hipMemcpyDeviceToHost);
    copy_array_transposed(q, device_addrs.q_dev, hipMemcpyDeviceToHost);

    gpuErrchk(hipFree(device_addrs.u_dev));
    gpuErrchk(hipFree(device_addrs.v_dev));
    gpuErrchk(hipFree(device_addrs.p_dev));
    gpuErrchk(hipFree(device_addrs.q_dev));
}

__global__ void col_sweep(int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q, DATA_TYPE a,
                          DATA_TYPE b, DATA_TYPE c, DATA_TYPE d, DATA_TYPE f) {
    for (int i = 1 + blockDim.x * blockIdx.x + threadIdx.x; i < _PB_N - 1; i += gridDim.x * blockDim.x) {
        DATA_TYPE prev_v = SCALAR_VAL(1.0);
        v[i * n + 0] = prev_v;
        DATA_TYPE prev_p = SCALAR_VAL(0.0);
        p[0 * n + i] = prev_p;
        DATA_TYPE prev_q = prev_v;
        q[0 * n + i] = prev_q;
        for (int j = 1; j < _PB_N - 1; j++) {
            double new_prev_p = -c / (a * prev_p + b);
            double new_prev_q = (-d * u[j * n + (i - 1)] + (SCALAR_VAL(1.0) + SCALAR_VAL(2.0) * d) * u[j * n + i] -
                                 f * u[j * n + (i + 1)] - a * prev_q) /
                                (a * prev_p + b);
            p[j * n + i] = new_prev_p;
            q[j * n + i] = new_prev_q;
            prev_p = new_prev_p;
            prev_q = new_prev_q;
        }

        prev_v = SCALAR_VAL(1.0);
        v[i * n + (_PB_N - 1)] = prev_v;
        for (int j = _PB_N - 2; j >= 1; j--) {
            prev_v = p[j * n + i] * prev_v + q[j * n + i];
            v[i * n + j] = prev_v;
        }
    }
}

__global__ void row_sweep(int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q, DATA_TYPE a,
                          DATA_TYPE c, DATA_TYPE d, DATA_TYPE e, DATA_TYPE f) {
    for (int i = 1 + blockDim.x * blockIdx.x + threadIdx.x; i < _PB_N - 1; i += gridDim.x * blockDim.x) {
        u[i * n + 0] = SCALAR_VAL(1.0);
        DATA_TYPE prev_p = SCALAR_VAL(0.0);
        p[0 * n + i] = prev_p;
        DATA_TYPE prev_q = u[i * n + 0];
        q[0 * n + i] = prev_q;
        for (int j = 1; j < _PB_N - 1; j++) {
            DATA_TYPE new_prev_p = -f / (d * prev_p + e);
            DATA_TYPE new_prev_q = (-a * v[j * n + (i - 1)] + (SCALAR_VAL(1.0) + SCALAR_VAL(2.0) * a) * v[j * n + i] -
                                    c * v[j * n + (i + 1)] - d * prev_q) /
                                   (d * prev_p + e);
            p[j * n + i] = new_prev_p;
            q[j * n + i] = new_prev_q;
            prev_p = new_prev_p;
            prev_q = new_prev_q;
        }
        DATA_TYPE prev_u = SCALAR_VAL(1.0);
        u[i * n + (_PB_N - 1)] = prev_u;
        for (int j = _PB_N - 2; j >= 1; j--) {
            prev_u = p[j * n + i] * prev_u + q[j * n + i];
            u[i * n + j] = prev_u;
        }
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
/* Based on a Fortran code fragment from Figure 5 of
 * "Automatic Data and Computation Decomposition on Distributed Memory Parallel
 * Computers" by Peizong Lee and Zvi Meir Kedem, TOPLAS, 2002
 */
void kernel_adi_inner(int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q) {
    DATA_TYPE DX, DY, DT;
    DATA_TYPE B1, B2;
    DATA_TYPE mul1, mul2;
    DATA_TYPE a, b, c, d, e, f;

    DX = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_N;
    DY = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_N;
    DT = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_TSTEPS;
    B1 = SCALAR_VAL(2.0);
    B2 = SCALAR_VAL(1.0);
    mul1 = B1 * DT / (DX * DX);
    mul2 = B2 * DT / (DY * DY);

    a = -mul1 / SCALAR_VAL(2.0);
    b = SCALAR_VAL(1.0) + mul1;
    c = a;
    d = -mul2 / SCALAR_VAL(2.0);
    e = SCALAR_VAL(1.0) + mul2;
    f = d;

    for (int t = 1; t <= _PB_TSTEPS; t++) {
        // Column Sweep
        col_sweep<<<32, 64>>>(tsteps, n, u, v, p, q, a, b, c, d, f);
        // Row Sweep
        row_sweep<<<32, 64>>>(tsteps, n, u, v, p, q, a, c, d, e, f);
    }
}

void kernel_adi(int tsteps, int n, DATA_TYPE POLYBENCH_2D(u, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(v, N2, N2, n, n),
                DATA_TYPE POLYBENCH_2D(p, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(q, N2, N2, n, n)) {
    kernel_adi_inner(tsteps, n, device_addrs.u_dev, device_addrs.v_dev, device_addrs.p_dev, device_addrs.q_dev);
    gpuErrchk(hipDeviceSynchronize());
}