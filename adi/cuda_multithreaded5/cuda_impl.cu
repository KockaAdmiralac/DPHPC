#include "hip/hip_runtime.h"
/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include <cuda_support.cuh>

#include "adi.h"

typedef struct {
    DATA_TYPE *u_dev;
    DATA_TYPE *v_dev;
    DATA_TYPE *p_dev;
    DATA_TYPE *q_dev;
} kernel_init_t;

kernel_init_t device_addrs;

void initialise_benchmark(int argc, char **argv, int tsteps, int n, DATA_TYPE POLYBENCH_2D(u, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_2D(v, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(p, N2, N2, n, n),
                          DATA_TYPE POLYBENCH_2D(q, N2, N2, n, n)) {
    (void)tsteps;
    (void)n;
    (void)argc;
    (void)argv;

    int i, j;

    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++) {
            u[i][j] = (DATA_TYPE)(i + n - j) / n;
        }

    gpuErrchk(hipMalloc(&device_addrs.u_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.v_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.p_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&device_addrs.q_dev, sizeof(DATA_TYPE) * n * n));

    gpuErrchk(hipMemcpy(device_addrs.u_dev, u, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.v_dev, v, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.p_dev, p, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_addrs.q_dev, q, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
}

void finish_benchmark(int tsteps, int n, DATA_TYPE POLYBENCH_2D(u, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_2D(v, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(p, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_2D(q, N2, N2, n, n)) {
    (void)tsteps;
    (void)n;
    gpuErrchk(hipMemcpy(u, device_addrs.u_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(v, device_addrs.v_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(p, device_addrs.p_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(q, device_addrs.q_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(device_addrs.u_dev));
    gpuErrchk(hipFree(device_addrs.v_dev));
    gpuErrchk(hipFree(device_addrs.p_dev));
    gpuErrchk(hipFree(device_addrs.q_dev));
}

__global__ void col_sweep(int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q, DATA_TYPE a,
                          DATA_TYPE b, DATA_TYPE c, DATA_TYPE d, DATA_TYPE f) {
    for (int i = 1 + blockDim.x * blockIdx.x + threadIdx.x; i < _PB_N - 1; i += gridDim.x * blockDim.x) {
        DATA_TYPE prev_v = SCALAR_VAL(1.0);
        v[0 * n + i] = prev_v;
        DATA_TYPE prev_p = SCALAR_VAL(0.0);
        p[i * n + 0] = prev_p;
        DATA_TYPE prev_q = prev_v;
        q[i * n + 0] = prev_q;
        for (int j = 1; j < _PB_N - 1; j++) {
            double new_prev_p = -c / (a * prev_p + b);
            double new_prev_q = (-d * u[j * n + (i - 1)] + (SCALAR_VAL(1.0) + SCALAR_VAL(2.0) * d) * u[j * n + i] -
                                 f * u[j * n + (i + 1)] - a * prev_q) /
                                (a * prev_p + b);
            p[i * n + j] = new_prev_p;
            q[i * n + j] = new_prev_q;
            v[j * n + i] = new_prev_q;  // writing to v here means penalty for cache, but accessing v (no bank
                                        // conflicts) is much better than q.
            prev_p = new_prev_p;
            prev_q = new_prev_q;
        }

        prev_v = SCALAR_VAL(1.0);
        v[(_PB_N - 1) * n + i] = prev_v;
        for (int j = _PB_N - 2; j >= 1; j--) {
            prev_v = p[i * n + j] * prev_v + v[j * n + i];
            v[j * n + i] = prev_v;
        }
    }
}

__global__ void row_sweep(int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q, DATA_TYPE a,
                          DATA_TYPE c, DATA_TYPE d, DATA_TYPE e, DATA_TYPE f) {
    for (int i = 1 + blockDim.x * blockIdx.x + threadIdx.x; i < _PB_N - 1; i += gridDim.x * blockDim.x) {
        u[i * n + 0] = SCALAR_VAL(1.0);
        DATA_TYPE prev_p = SCALAR_VAL(0.0);
        p[i * n + 0] = prev_p;
        DATA_TYPE prev_q = u[i * n + 0];
        q[i * n + 0] = prev_q;
        for (int j = 1; j < _PB_N - 1; j++) {
            DATA_TYPE new_prev_p = -f / (d * prev_p + e);
            DATA_TYPE new_prev_q = (-a * v[(i - 1) * n + j] + (SCALAR_VAL(1.0) + SCALAR_VAL(2.0) * a) * v[i * n + j] -
                                    c * v[(i + 1) * n + j] - d * prev_q) /
                                   (d * prev_p + e);
            p[i * n + j] = new_prev_p;
            q[i * n + j] = new_prev_q;
            prev_p = new_prev_p;
            prev_q = new_prev_q;
        }
        DATA_TYPE prev_u = SCALAR_VAL(1.0);
        u[i * n + (_PB_N - 1)] = prev_u;
        for (int j = _PB_N - 2; j >= 1; j--) {
            prev_u = p[i * n + j] * prev_u + q[i * n + j];
            u[i * n + j] = prev_u;
        }
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
/* Based on a Fortran code fragment from Figure 5 of
 * "Automatic Data and Computation Decomposition on Distributed Memory Parallel
 * Computers" by Peizong Lee and Zvi Meir Kedem, TOPLAS, 2002
 */
void kernel_adi_inner(int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q) {
    DATA_TYPE DX, DY, DT;
    DATA_TYPE B1, B2;
    DATA_TYPE mul1, mul2;
    DATA_TYPE a, b, c, d, e, f;

    DX = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_N;
    DY = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_N;
    DT = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_TSTEPS;
    B1 = SCALAR_VAL(2.0);
    B2 = SCALAR_VAL(1.0);
    mul1 = B1 * DT / (DX * DX);
    mul2 = B2 * DT / (DY * DY);

    a = -mul1 / SCALAR_VAL(2.0);
    b = SCALAR_VAL(1.0) + mul1;
    c = a;
    d = -mul2 / SCALAR_VAL(2.0);
    e = SCALAR_VAL(1.0) + mul2;
    f = d;

    for (int t = 1; t <= _PB_TSTEPS; t++) {
        // Column Sweep
        col_sweep<<<32, 64>>>(tsteps, n, u, v, p, q, a, b, c, d, f);
        // Row Sweep
        row_sweep<<<32, 64>>>(tsteps, n, u, v, p, q, a, c, d, e, f);
    }
}

void kernel_adi(int tsteps, int n, DATA_TYPE POLYBENCH_2D(u, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(v, N2, N2, n, n),
                DATA_TYPE POLYBENCH_2D(p, N2, N2, n, n), DATA_TYPE POLYBENCH_2D(q, N2, N2, n, n)) {
    kernel_adi_inner(tsteps, n, device_addrs.u_dev, device_addrs.v_dev, device_addrs.p_dev, device_addrs.q_dev);
    gpuErrchk(hipDeviceSynchronize());
}
