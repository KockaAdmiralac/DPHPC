#include "hip/hip_runtime.h"
/* Include polybench common header. */
#include <hipblas.h>
#include <polybench.h>
#include <stdbool.h>

/* Include benchmark-specific header. */
#include <hip/hip_runtime_api.h>

#include <cuda_support.cuh>

#include "adi.h"

typedef struct {
    default_adi_data_t adi_data;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    DATA_TYPE *u_dev;
    DATA_TYPE *v_dev;
    DATA_TYPE *p_dev;
    DATA_TYPE *q_dev;
    DATA_TYPE *spare_arr;
    bool u_transposed;
    bool v_transposed;
} cuda_adi_data_t;

void transpose_in_place(DATA_TYPE POLYBENCH_2D(arr, N2, N2, n, n)) {
    for (int i = 0; i < N2; i++) {
        for (int j = i + 1; j < N2; j++) {
            DATA_TYPE upper_right = arr[i][j];
            DATA_TYPE lower_left = arr[j][i];
            arr[j][i] = upper_right;
            arr[i][j] = lower_left;
        }
    }
}

inline void transpose_oop(hipblasHandle_t handle, double *src_arr, double *dst_arr, int n) {
    double c1 = 1;
    double c0 = 0;
    gpuCublasErrchk(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &c1, src_arr, n, &c0, src_arr, n, dst_arr, n));
}

__global__ void copy_array(double *src_arr, double *dst_arr, int n, int row_count) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n * row_count; i += blockDim.x) {
        dst_arr[i] = src_arr[i];
    }
}

void copy_array_transposed(DATA_TYPE POLYBENCH_2D(arr, N2, N2, n, n), double *device_addr, hipMemcpyKind dir) {
    if (dir == hipMemcpyHostToDevice) {
        transpose_in_place(arr);
        gpuErrchk(hipMemcpy(device_addr, arr, sizeof(DATA_TYPE) * N2 * N2, hipMemcpyHostToDevice));
    } else {
        gpuErrchk(hipMemcpy(arr, device_addr, sizeof(DATA_TYPE) * N2 * N2, hipMemcpyDeviceToHost));
        gpuErrchk(hipDeviceSynchronize());
        transpose_in_place(arr);
    }
}

void initialise_benchmark(int argc, char **argv, int tsteps, int n, void **gen_data_ptr) {
    *gen_data_ptr = polybench_alloc_data(1, sizeof(cuda_adi_data_t));
    cuda_adi_data_t *data_ptr = (cuda_adi_data_t *)(*gen_data_ptr);
    default_initialise_benchmark(argc, argv, tsteps, n, &data_ptr->adi_data);

    gpuErrchk(hipMalloc(&data_ptr->u_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&data_ptr->v_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&data_ptr->p_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&data_ptr->q_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&data_ptr->spare_arr, sizeof(DATA_TYPE) * n * n));

    data_ptr->u_transposed = true;
    data_ptr->v_transposed = true;

    gpuCublasErrchk(hipblasCreate(&data_ptr->handle));

    gpuErrchk(hipMemcpy(data_ptr->u_dev, data_ptr->adi_data.u, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice));
    copy_array_transposed(data_ptr->adi_data.v, data_ptr->v_dev, hipMemcpyHostToDevice);
    copy_array_transposed(data_ptr->adi_data.p, data_ptr->p_dev, hipMemcpyHostToDevice);
    copy_array_transposed(data_ptr->adi_data.q, data_ptr->q_dev, hipMemcpyHostToDevice);
    gpuErrchk(hipDeviceSynchronize());
    hipProfilerStart();
}

void finish_benchmark(void *gen_data_ptr) {
    hipProfilerStop();
    cuda_adi_data_t *data_ptr = (cuda_adi_data_t *)gen_data_ptr;
    int n = data_ptr->adi_data.n;
    gpuErrchk(hipMemcpy(data_ptr->adi_data.u, data_ptr->u_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost));
    copy_array_transposed(data_ptr->adi_data.v, data_ptr->v_dev, hipMemcpyDeviceToHost);
    copy_array_transposed(data_ptr->adi_data.p, data_ptr->p_dev, hipMemcpyDeviceToHost);
    copy_array_transposed(data_ptr->adi_data.q, data_ptr->q_dev, hipMemcpyDeviceToHost);
}

void print_data(int argc, char **argv, int n, void *gen_data_ptr) {
    default_print_data(argc, argv, n, &((cuda_adi_data_t *)gen_data_ptr)->adi_data);
}

void free_data(void *gen_data_ptr) {
    cuda_adi_data_t *data_ptr = (cuda_adi_data_t *)gen_data_ptr;
    gpuErrchk(hipFree(data_ptr->u_dev));
    gpuErrchk(hipFree(data_ptr->v_dev));
    gpuErrchk(hipFree(data_ptr->p_dev));
    gpuErrchk(hipFree(data_ptr->q_dev));
    gpuErrchk(hipFree(data_ptr->spare_arr));
    gpuCublasErrchk(hipblasDestroy(data_ptr->handle));
    free(gen_data_ptr);
}

__global__ void transpose_row_to_column(double *out_arr, double *in_arr, int n, int src_row) {
    for (int src_col = blockDim.x * blockIdx.x + threadIdx.x; src_col < n; src_col += blockDim.x) {
        out_arr[src_col * n + src_row] = in_arr[src_row * n + src_col];
    }
}

__global__ void copy_row(double *out_arr, double *in_arr, int n, int src_row) {
    for (int src_col = blockDim.x * blockIdx.x + threadIdx.x; src_col < n; src_col += blockDim.x) {
        out_arr[src_row * n + src_col] = in_arr[src_row * n + src_col];
    }
}

inline __device__ int get_idx(int j, int n, int i, bool jni) {
    if (jni) {
        return j * n + i;
    } else {
        return i * n + j;
    }
}
// #define PUT_IN_ARR_IN_SM

#define SWEEP_TPB 1024

__device__ void sweep_generic(int n, const double *in_arr, double *out_arr, double *p, double *q, double v1, double v2,
                              double v3, double v4, double v5, bool in_jni, bool out_jni) {
    // col_sweep: in_arr = u, out_arr = v, v1=a, v2=b, v3=c, v4=d, v5=f
    // row_sweep: in_arr = v, out_arr = u, v1=d, v2=e, v3=f, v4=a, v5=c

    /*
    min bandwidth estimates per timestep:
    load in_arr (N^2)
    store p and q (each N^2)
    load p and q (each N^2)
    store out_arr (N^2)
    */

#ifdef PUT_IN_ARR_IN_SM
    const int threads_in_block = SWEEP_TPB;
    const int threads_in_block_plus_boundaries = threads_in_block;
    const int total_smem_slots = 48 * 1024 / sizeof(DATA_TYPE);
    const int js_at_time = total_smem_slots / threads_in_block_plus_boundaries;
    __shared__ DATA_TYPE smem_in_arr[threads_in_block_plus_boundaries * js_at_time];
#endif

    int base_iter = 1;
    int end_iter_excl = _PB_N - 1;
    for (int i = base_iter + blockDim.x * blockIdx.x + threadIdx.x; i < end_iter_excl; i += gridDim.x * blockDim.x) {
#ifdef PUT_IN_ARR_IN_SM
        int block_start_i = i - ((i - base_iter) % SWEEP_TPB);
        int block_end_i = block_start_i + SWEEP_TPB;
        int active_threads_this_block = block_end_i >= end_iter_excl ? end_iter_excl - block_start_i : SWEEP_TPB;
#endif

        DATA_TYPE prev_p = SCALAR_VAL(0.0);
        p[0 * n + i] = prev_p;
        DATA_TYPE prev_q = SCALAR_VAL(1.0);
        q[0 * n + i] = prev_q;

        // copy in_arr for next few iterations into smem

        const int init_j = 1;
#ifdef PUT_IN_ARR_IN_SM
        int j_ctr = js_at_time;
#endif
        for (int j = init_j; j < _PB_N - 1; j++) {
#ifdef PUT_IN_ARR_IN_SM
#define SMEM_ADDR(thread, j_temp) ((thread) * js_at_time + (j_temp))
            // #define SMEM_ADDR(thread, j_temp) ((j_temp) * SWEEP_TPB + (thread))
            if (j_ctr == js_at_time) {
                const int js_to_load = _PB_N - 1 - j >= js_at_time ? js_at_time : _PB_N - 1 - j;
                __syncthreads();  // avoid overwriting a slowpoke's shared memory
                for (int j_temp = 0; j_temp < js_to_load && j_temp < js_at_time; j_temp++) {
                    // smem_in_arr[j_temp * SWEEP_TPB + threadIdx.x] =
                    smem_in_arr[SMEM_ADDR(threadIdx.x, j_temp)] =
                        in_arr[get_idx(j + j_temp, n, i, in_jni)];  // note this i is >= 1, not >= 0
                }
                __syncthreads();  // needed to ensure my neighbour threads don't go ahead until my data for them is
                                  //   available.
                j_ctr = 0;
            }
#endif
            DATA_TYPE new_prev_p = -v3 / (v1 * prev_p + v2);
#ifdef PUT_IN_ARR_IN_SM
            // only edges of block will use global memory because it simplifies storage into shared memory
            DATA_TYPE inm1 = threadIdx.x == 0 ? in_arr[get_idx(j, n, i - 1, in_jni)]
                                              : smem_in_arr[SMEM_ADDR(threadIdx.x - 1, j_ctr)];
            DATA_TYPE in0 = smem_in_arr[SMEM_ADDR(threadIdx.x, j_ctr)];
            DATA_TYPE inp1 = threadIdx.x == active_threads_this_block - 1
                                 ? in_arr[get_idx(j, n, i + 1, in_jni)]
                                 : smem_in_arr[SMEM_ADDR(threadIdx.x + 1, j_ctr)];
#else
            DATA_TYPE in0 = in_arr[get_idx(j, n, i, in_jni)];
            DATA_TYPE inm1 = in_arr[get_idx(j, n, i - 1, in_jni)];
            DATA_TYPE inp1 = in_arr[get_idx(j, n, i + 1, in_jni)];
#endif

            DATA_TYPE new_prev_q =
                (-v4 * inm1 + (SCALAR_VAL(1.0) + SCALAR_VAL(2.0) * v4) * in0 - v5 * inp1 - v1 * prev_q) /
                (v1 * prev_p + v2);

            p[j * n + i] = new_prev_p;
            q[j * n + i] = new_prev_q;
            prev_p = new_prev_p;
            prev_q = new_prev_q;

#ifdef PUT_IN_ARR_IN_SM
            j_ctr++;
#endif
        }

        DATA_TYPE prev_out = SCALAR_VAL(1.0);
        out_arr[get_idx(0, n, i, out_jni)] = SCALAR_VAL(1.0);
        for (int j = _PB_N - 2; j >= 1; j--) {
            prev_out = p[j * n + i] * prev_out + q[j * n + i];
            out_arr[get_idx(j, n, i, out_jni)] = prev_out;
        }
        out_arr[get_idx(_PB_N - 1, n, i, out_jni)] = SCALAR_VAL(1.0);
    }
}

__global__ void col_sweep(int tsteps, int n, const DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q, DATA_TYPE a,
                          DATA_TYPE b, DATA_TYPE c, DATA_TYPE d, DATA_TYPE f, bool in_jni, bool out_jni) {
    sweep_generic(n, u, v, p, q, a, b, c, d, f, in_jni, out_jni);
}

__global__ void row_sweep(int tsteps, int n, DATA_TYPE *u, const DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q, DATA_TYPE a,
                          DATA_TYPE c, DATA_TYPE d, DATA_TYPE e, DATA_TYPE f, bool in_jni, bool out_jni) {
    sweep_generic(n, v, u, p, q, d, e, f, a, c, in_jni, out_jni);
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
/* Based on a Fortran code fragment from Figure 5 of
 * "Automatic Data and Computation Decomposition on Distributed Memory Parallel
 * Computers" by Peizong Lee and Zvi Meir Kedem, TOPLAS, 2002
 */
void kernel_adi_inner(hipblasHandle_t handle, int tsteps, int n, DATA_TYPE *u, DATA_TYPE *v, DATA_TYPE *p, DATA_TYPE *q,
                      DATA_TYPE *spare_arr, bool u_transposed, bool v_transposed) {
    DATA_TYPE DX, DY, DT;
    DATA_TYPE B1, B2;
    DATA_TYPE mul1, mul2;
    DATA_TYPE a, b, c, d, e, f;

    DX = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_N;
    DY = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_N;
    DT = SCALAR_VAL(1.0) / (DATA_TYPE)_PB_TSTEPS;
    B1 = SCALAR_VAL(2.0);
    B2 = SCALAR_VAL(1.0);
    mul1 = B1 * DT / (DX * DX);
    mul2 = B2 * DT / (DY * DY);

    a = -mul1 / SCALAR_VAL(2.0);
    b = SCALAR_VAL(1.0) + mul1;
    c = a;
    d = -mul2 / SCALAR_VAL(2.0);
    e = SCALAR_VAL(1.0) + mul2;
    f = d;

    int sms = get_device_multiprocessors(0);

    // should be able to zero out only once during program run

    for (int t = 1; t <= _PB_TSTEPS; t++) {
        // Column Sweep
        // Using very high block/thread count to give CUDA lots of independent work
        // threads have negligible memory needs so 1024/block makes sense.
        double *temp_v = spare_arr;
        if (v_transposed) {
        transpose_row_to_column<<<sms, 1024>>>(temp_v, v, n, 0);
        transpose_row_to_column<<<sms, 1024>>>(temp_v, v, n, _PB_N - 1);
        }
        col_sweep<<<sms, SWEEP_TPB>>>(tsteps, n, u, temp_v, p, q, a, b, c, d, f, true, v_transposed);
        if (v_transposed) {
        transpose_oop(handle, temp_v, v, n);
        } else {
            copy_array<<<sms, 1024>>>(&temp_v[n], &v[n], n, n-2);
        }

        // Row Sweep
        double *temp_u = spare_arr;
        if (u_transposed) {
            transpose_row_to_column<<<sms, 1024>>>(temp_u, u, n, 0);
            transpose_row_to_column<<<sms, 1024>>>(temp_u, u, n, _PB_N - 1);
        }
        row_sweep<<<sms, SWEEP_TPB>>>(tsteps, n, temp_u, v, p, q, a, c, d, e, f, true, u_transposed);
        if (u_transposed) {
            transpose_oop(handle, temp_u, u, n);
        } else {
            copy_array<<<sms, 1024>>>(&temp_u[n], &u[n], n, n-2);
        }
    }
}

void kernel_adi(void *gen_data_ptr) {
    cuda_adi_data_t *data_ptr = (cuda_adi_data_t *)gen_data_ptr;
    kernel_adi_inner(data_ptr->handle, data_ptr->adi_data.tsteps, data_ptr->adi_data.n, data_ptr->u_dev,
                     data_ptr->v_dev, data_ptr->p_dev, data_ptr->q_dev, data_ptr->spare_arr, data_ptr->u_transposed, data_ptr->v_transposed);
    gpuErrchk(hipDeviceSynchronize());
}
