#include "hip/hip_runtime.h"
#include <assert.h>
#include <gemver.h>
#include <polybench.h>
#include <stdio.h>

#include <cuda_support.cuh>

typedef struct {
    DATA_TYPE *A_dev = NULL;
    DATA_TYPE *u1_dev = NULL;
    DATA_TYPE *v1_dev = NULL;
    DATA_TYPE *u2_dev = NULL;
    DATA_TYPE *v2_dev = NULL;
    DATA_TYPE *w_dev = NULL;
    DATA_TYPE *x_dev = NULL;
    DATA_TYPE *y_dev = NULL;
    DATA_TYPE *z_dev = NULL;
} kernel_data_t;

kernel_data_t kdata;

void initialise_benchmark(int argc, char **argv, int n, DATA_TYPE *alpha, DATA_TYPE *beta,
                          DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n), DATA_TYPE POLYBENCH_1D(u1, N2, n),
                          DATA_TYPE POLYBENCH_1D(v1, N2, n), DATA_TYPE POLYBENCH_1D(u2, N2, n),
                          DATA_TYPE POLYBENCH_1D(v2, N2, n), DATA_TYPE POLYBENCH_1D(w, N2, n),
                          DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                          DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    *alpha = 1.5;
    *beta = 1.2;
    (void)argc;
    (void)argv;

    DATA_TYPE fn = (DATA_TYPE)n;

    for (int i = 0; i < n; i++) {
        u1[i] = i;
        u2[i] = ((i + 1) / fn) / 2.0;
        v1[i] = ((i + 1) / fn) / 4.0;
        v2[i] = ((i + 1) / fn) / 6.0;
        y[i] = ((i + 1) / fn) / 8.0;
        z[i] = ((i + 1) / fn) / 9.0;
        x[i] = 0.0;
        w[i] = 0.0;
        for (int j = 0; j < n; j++) A[i][j] = (DATA_TYPE)(i * j % n) / n;
    }

    gpuErrchk(hipMalloc(&kdata.A_dev, sizeof(DATA_TYPE) * n * n));
    gpuErrchk(hipMalloc(&kdata.u1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.v1_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.u2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.v2_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.w_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.x_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.y_dev, sizeof(DATA_TYPE) * n));
    gpuErrchk(hipMalloc(&kdata.z_dev, sizeof(DATA_TYPE) * n));

    hipMemcpy(kdata.A_dev, A, sizeof(DATA_TYPE) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.u1_dev, u1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.v1_dev, v1, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.u2_dev, u2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.v2_dev, v2, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.w_dev, w, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.x_dev, x, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.y_dev, y, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);
    hipMemcpy(kdata.z_dev, z, sizeof(DATA_TYPE) * n, hipMemcpyHostToDevice);

    gpuErrchk(hipDeviceSynchronize());
}

void finish_benchmark(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                      DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                      DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                      DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n),
                      DATA_TYPE POLYBENCH_1D(y, N2, n), DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    (void)alpha;
    (void)beta;
    (void)u1;
    (void)v1;
    (void)u2;
    (void)v2;
    (void)w;
    (void)y;
    (void)z;

    hipMemcpy(w, kdata.w_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);
    hipMemcpy(A, kdata.A_dev, sizeof(DATA_TYPE) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(x, kdata.x_dev, sizeof(DATA_TYPE) * n, hipMemcpyDeviceToHost);

    hipFree(kdata.A_dev);
    hipFree(kdata.u1_dev);
    hipFree(kdata.v1_dev);
    hipFree(kdata.u2_dev);
    hipFree(kdata.v2_dev);
    hipFree(kdata.w_dev);
    hipFree(kdata.x_dev);
    hipFree(kdata.y_dev);
    hipFree(kdata.z_dev);
}

__global__ void kernel_3(const int n, const DATA_TYPE alpha, const DATA_TYPE *A, DATA_TYPE *w, const DATA_TYPE *x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < _PB_N; i += gridDim.x * blockDim.x) {
        DATA_TYPE loc = w[i];
        for (int j = 0; j < _PB_N; j++) {
            int jmod = (j + threadIdx.x % 32) % _PB_N;
            loc += alpha * A[i * n + jmod] * x[jmod];
        }
        w[i] = loc;
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
void kernel_gemver(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N2, N2, n, n),
                   DATA_TYPE POLYBENCH_1D(u1, N2, n), DATA_TYPE POLYBENCH_1D(v1, N2, n),
                   DATA_TYPE POLYBENCH_1D(u2, N2, n), DATA_TYPE POLYBENCH_1D(v2, N2, n),
                   DATA_TYPE POLYBENCH_1D(w, N2, n), DATA_TYPE POLYBENCH_1D(x, N2, n), DATA_TYPE POLYBENCH_1D(y, N2, n),
                   DATA_TYPE POLYBENCH_1D(z, N2, n)) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int sm_count = deviceProp.multiProcessorCount;
    int max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;
    int max_threads_for_gpu = sm_count * max_threads_per_sm;

    dim3 tpb1(64);
    int blocks_k1 = max_threads_for_gpu / tpb1.x;
    dim3 bpg1(blocks_k1);
    // printf("Using %d blocks_k1, max threads for GPU is %d, max threads per SM is %d\n", blocks_k1,
    // max_threads_for_gpu, max_threads_per_sm);

    dim3 tpb2(64);
    dim3 bpg2(max_threads_for_gpu / tpb2.x);
    dim3 tpb3(64);
    dim3 bpg3(max_threads_for_gpu / tpb3.x);
    kernel_3<<<bpg3, tpb3>>>(n, alpha, kdata.A_dev, kdata.w_dev, kdata.x_dev);

    hipDeviceSynchronize();
}
